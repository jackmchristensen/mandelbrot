#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "../include/mandelbrot.cuh"

namespace cuda {
  hipGraphicsResource* cuda_pbo;
}

__global__ void gradient(uchar4* pixels, int width, int height) {

}

void registerPixelBuffer(GLuint pbo) {
  hipGraphicsGLRegisterBuffer(&cuda::cuda_pbo, pbo, cudaGraphicsMapFlagsWriteDiscard);
}

void drawGradient(int width, int height) {
  hipGraphicsMapResources(1, &cuda::cuda_pbo);
  uchar4* d_pixels = nullptr;
  size_t bytes = 0;
  hipGraphicsResourceGetMappedPointer((void**)&d_pixels, &bytes, cuda::cuda_pbo);

  dim3 block(16, 16);
  dim3 grid((width+block.x-1)/block.x, (height+block.y-1)/block.y);
  gradient<<<block, grid>>>(d_pixels, width, height);

  hipGraphicsUnmapResources(1, &cuda::cuda_pbo);
}
